
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *res, int a, int b) {
    *res = a + b;
}

int main(void) {
    int res;
    int *device_res = NULL;

    hipError_t mres;
    // Allocate memory on the device
    // You cannot dereference this in host code!
    mres = hipMalloc(&device_res, sizeof(int));
    if (mres != hipSuccess) {
        printf("Malloc failed\n");
        return -1;
    }
    // Do computation
    add<<<1,1>>>(device_res, 2, 7);
    // Copy result back to host
    mres = hipMemcpy(&res, device_res, sizeof(int), hipMemcpyDeviceToHost);
    if (mres != hipSuccess) {
        printf("Memcpy failed\n");
        return -1;
    }

    // Now free the memory we allocated on the device
    mres = hipFree(device_res);
    if (mres != hipSuccess) {
        printf("Free failed\n");
        return -1;
    }

    printf("2 + 7 = %d\n", res);
    return 0;
}

